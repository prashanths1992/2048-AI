#include "hip/hip_runtime.h"
#include <iostream>
#include<math.h>
#include<queue>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include<hip/hip_runtime_api.h>

using namespace std;


void __global__ moveLeft(int* matrix)
{

    int mergeCount;
    int temp[4];
    
    //printf("Hello from GPU..\n");
    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	
    while (1) {
		//printf("Running Left");
        mergeCount = 0;
		printf("Left Merge Count before: %d \n",mergeCount);
		
        for (int i = 4*(threadIdx.x+blockIdx.x*blockDim.x)+1; i < 4*(threadIdx.x+blockIdx.x*blockDim.x)+4; i++) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 1] && temp[i - 1] != 1 && temp[i] != 1) {
                matrix[i - 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[i - 1] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 1] == 0) {
                matrix[i - 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
        if (mergeCount == 0) break;

    }

}


void __global__ moveRight(int* matrix)
{

    int mergeCount;
    int temp[4];

    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	int start=4*(threadIdx.x+blockIdx.x*blockDim.x)+2;
	int end=4*(threadIdx.x+blockIdx.x*blockDim.x);
	
	//printf("Thread Idx: %d , start: %d , end: %d", threadIdx.x,4*(threadIdx.x+blockIdx.x*blockDim.x)+2,4*(threadIdx.x+blockIdx.x*blockDim.x));
    while (1) {
		printf("Running Right");
        mergeCount = 0;
		//printf("Merge Count before: %d \n",mergeCount);
        for (int i = start; i >=end; i--) {
			//printf("Running Right for loop %d  %d  %d %d",i,(threadIdx.x+blockIdx.x*blockDim.x),blockIdx.x,blockDim.x);
            if (matrix[i] != 0 && matrix[i] == matrix[i + 1] && temp[i + 1] != 1 && temp[i] != 1) {
                matrix[i + 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[i + 1] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 1] == 0) {
                matrix[i + 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
		//printf("Merge Count outside for: %d \n",mergeCount);
        if (mergeCount == 0) break;
		break;

    }
}


void __global__ moveUp(int* matrix)
{

    int mergeCount;
    int temp[16];
    
    for(int i=0;i<16;i++){
        temp[i]=0;
    }

    while (1) {

        mergeCount = 0;

        for (int i = (threadIdx.x+blockIdx.x*16)+4; i < (blockIdx.x+1)*16; i+=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 4] && temp[(i - 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i - 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i - 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 4] == 0) {
                matrix[i - 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
//               mergeCount--;
        }

        if (mergeCount == 0) break;

    }
}


void __global__ moveDown(int* matrix)
{

    int mergeCount;
    int temp[16];

    int start=((blockIdx.x+1)*16)-8+threadIdx.x;
	int end = (blockIdx.x)*16;
	//printf("STrt: %d End: %d",start,end);
    while (1) {

        mergeCount = 0;

        for (int i = start; i >= end ; i-=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i + 4] && temp[(i + 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i + 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i + 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 4] == 0) {
                matrix[i + 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
//               mergeCount--;
        }

        if (mergeCount == 0) break;

    }
}


void printMatrix(int*** matrix)
{
    for(int i=0;i<4;i++) {
        for (int j = 0; j < 4; j++) {

            if(matrix[i][j]!=0)
                cout<<pow(2,matrix[0][i][j])<<" ";
            else
                cout<<0<<" ";

        }
        cout<<endl;
    }
}

void __global__ generate_copy_for_child_type1(int* input_matrix,int* output_matrix, int* output_rv, int n){
	  int thread_index = threadIdx.x;
	  int block_index = blockIdx.x;
	  int start=((block_index)*16);
      for(int child=0;child<4;child++){
          int child_start = ((blockIdx.x*4)+child)*16;
          output_matrix[child_start+thread_index]=input_matrix[start+thread_index];
      }
      output_rv[block_index]=0;
}

void __global__ consolidate_return_values(int* input_rv,int* output_rv,int n){
		int block_index = blockIdx.x;
      int i = 4*block_index;
      output_rv[block_index] = max(max(input_rv[i],input_rv[i+1]),max(input_rv[i+2],input_rv[i+3]));
}

__global__ void runExpectiMaxSearch(int* matrix,int n,int type,int level,int* rv)
{

    int max_level = 5;
    if(level == max_level)
    {
        printf("Terminal.........\n");
        int i=threadIdx.x+(blockDim.x*blockIdx.x);
        if(i<n){
            rv[i]=0;
            int index_i = i*16;
            for(int j=0;j<4;j++){
                int index_j = j*4;
                for(int k=0;k<4;k++){
                    if(matrix[index_i+index_j+k]==0){
                        rv[i]++;
                    }
                }
            }
        }
    }

    else if(type == 1)
    {
        //LRUD configuration
        printf("Directions.........\n");
        int* child_matrix = (int *)malloc(n*4*16*sizeof(int));
        int* child_rv = (int *)malloc(n*4*sizeof(int));
        generate_copy_for_child_type1<<<n,16>>>((int *)matrix,child_matrix,child_rv,n);
        hipDeviceSynchronize();

		printf("1.........\n");
        hipStream_t left;
        hipStreamCreateWithFlags(&left, hipStreamNonBlocking);
        moveLeft<<<n,4,0,left>>>(child_matrix);
        
		hipDeviceSynchronize();
		printf("1.1.........\n");
		
        hipStream_t right;
        hipStreamCreateWithFlags(&right, hipStreamNonBlocking);
        moveRight<<<n,4,0,right>>>(child_matrix);
        
		hipDeviceSynchronize();
		printf("1.2.........\n");
        hipStream_t up;
        hipStreamCreateWithFlags(&up, hipStreamNonBlocking);
        moveUp<<<n,4,0,up>>>(child_matrix);
        
		hipDeviceSynchronize();
		printf("1.3.........\n");
        hipStream_t down;
        hipStreamCreateWithFlags(&down, hipStreamNonBlocking);
        moveDown<<<n,4,0,down>>>(child_matrix);
        hipDeviceSynchronize();
		printf("2.........\n");

		if(level!=max_level-1){
			runExpectiMaxSearch<<<1,1>>>(child_matrix,4*n,type+1,level+1,child_rv);
		}
		else if(level==max_level-1){
			runExpectiMaxSearch<<<1,1>>>(child_matrix,4*n,type+1,level+1,child_rv);
		}
        hipDeviceSynchronize();

		printf("3.........\n");
        consolidate_return_values<<<n,1>>>(child_rv,rv,n);
        hipDeviceSynchronize();
        printf("4.........\n");
        if(level==0){
            if(child_rv[0]>child_rv[1]){
                if(child_rv[0]>child_rv[2]){
                    if(child_rv[0]>child_rv[3]){
                        rv[0]=0;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
            else{
                if(child_rv[1]>child_rv[2]){
                    if(child_rv[1]>child_rv[3]){
                        rv[0]=1;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
        }
        free(child_rv);
		free(child_matrix);
		printf("5.........\n");
    }
    else {
        printf("Chances.........\n");
        int count = 0;
		int *n_count;
		n_count = (int *)malloc(sizeof(int)*n);
        int *temp;
        temp = (int *)malloc(sizeof(int)*n);

        for (int i = 0; i < n; i++) {
			int index_i = i*16;
			int num=0;
			temp[i]=0;
			n_count[i]=0;
            for (int j = 0; j < 4; j++) {
				int index_j = j*4;
                for (int k = 0; k < 4; k++) {
                    if (matrix[index_i+index_j+k] == 0){
                        count++;
						num++;
					}
                }
            }

            if (i == 0){
                temp[0] = count;
				n_count[0] = count;
			}
            else{
                temp[i] = num;
				n_count[i] = count;
			}

        }

        int *matrix1 = (int *)malloc(count*16*(sizeof(int)));

        int *rv1 = (int *)malloc(count*sizeof(int));

        for (int i = 0; i < count; i++) rv1[i] = 0;

		int i=0;
		int k=0;
		
        for (int l = 0; l < n; l++) {
            int flag=0;
			int index_l = l*16;
            for (int i = 0; i < temp[l]; i++) {
                int index = i;
                if (l != 0) {
                    index += n_count[l - 1];
                }
                flag=0;
				int index_i = index*16;
                for (int j = 0; j < 4; j++) {
					int index_j = j*4;
                    for (int k = 0; k < 4; k++) {
                        if (matrix[index_l+index_j+k] == 0 && flag==0) {
                            matrix1[index_i+index_j+k] = 1;
                            matrix[index_l+index_j+k] = -1;
                            flag==1;
                        } else {
                            matrix1[index_i+index_j+k] = matrix[index_l+index_j+k];
                            if(matrix[index_l+index_j+k] == -1){
                                matrix1[index_i+index_j+k] == 0;
                            }
                        }

                    }
                }

            }

        }

		if(level!=max_level-1){
			runExpectiMaxSearch<<<1,1>>>(matrix1, count, type-1, level + 1, rv1);
		}
		else if(level==max_level-1){
			runExpectiMaxSearch<<<1,count>>>(matrix1, count, type-1, level + 1, rv1);
		}

		hipDeviceSynchronize();
		int l,p;
		l=k=p=0;
        for (int l = 0; l < n; l++) {
            int q= temp[l];
			if(q==0){
				rv[k]=0;
				k++;
				continue;
			}
			while(q--){
				rv[k]+=rv1[p++];
			}
			rv[k]/=temp[l];
			k++;
        }
		free(matrix1);
		free(temp);
		free(n_count);
		free(rv1);
    }

}

int main() {

    int* matrix = (int *)malloc(sizeof(int)*16);
	int* d_matrix;
    int* rv = (int *)malloc(sizeof(int));
	int *d_rv;

	int matrix_size = sizeof(int)*16;
	int rv_size = sizeof(int)*1;
	hipMalloc(&d_matrix,matrix_size);
	hipMalloc(&d_rv,rv_size);
	
    rv[0]=0;


    matrix[0] = 2;
    matrix[1] = 2;
    matrix[2] = 3;
    matrix[3] = 4;

    matrix[4] = 2;
    matrix[5] = 2;
    matrix[6] = 0;
    matrix[7] = 0;

    matrix[8] = 2;
    matrix[9] = 0;
    matrix[10] = 0;
    matrix[11] = 0;

    matrix[12] = 3;
    matrix[13] = 4;
    matrix[14] = 4;
    matrix[15] = 5;
	
	
    cout<<"Boooooom"<<endl;
    
	for(int i=0;i<16;i++){
		cout<<matrix[i]<<" ";
	}
	cout<<endl;
	hipMemcpy(d_matrix,matrix,matrix_size,hipMemcpyHostToDevice);
	hipMemcpy(d_rv,rv,rv_size,hipMemcpyHostToDevice);
	
	runExpectiMaxSearch<<<1,1>>>(d_matrix,1,1,0,d_rv);
	
	hipDeviceSynchronize();
	hipError_t err;
	err = hipGetLastError();
	if(err!=hipSuccess){
		printf("Error %s \n",hipGetErrorString(err));
	}
	//hipMemcpy(matrix,d_matrix,matrix_size,hipMemcpyDeviceToHost);
	hipMemcpy(rv,d_rv,rv_size,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(int i=0;i<16;i++){
		cout<<matrix[i]<<" ";
	}
	cout<<endl;

    if(rv[0]==0){
        cout<<"Left"<<endl;
    }
    if(rv[0]==1){
        cout<<"Right"<<endl;
    }
    if(rv[0]==2){
        cout<<"Up"<<endl;
    }
    if(rv[0]==3){
        cout<<"Down"<<endl;
    }

    return 0;
}

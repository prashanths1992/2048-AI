#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <queue>
#include <sys/types.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

string reset = "\u001b[0m";

unsigned GetNumberOfDigits (int i);

string getColor(int value)
{
    switch(value%10)
    {

        case 1:
            return "\u001b[37m\u001b[43;1m";

        case 2:
            return "\u001b[37m\u001b[44;1m";

        case 3:
            return "\u001b[37m\u001b[45;1m";

        case 4:
            return "\u001b[37m\u001b[46;1m";

        case 5:
            return "\u001b[37m\u001b[40;1m";

        case 6:
            return "\u001b[37m\u001b[41m";

        case 7:
            return "\u001b[37m\u001b[41;1m";

        case 8:
            return "\u001b[37m\u001b[42;1m";

        case 9:
            return "\u001b[37m\u001b[42m";

//        case 0:
//            return "\u001b[37m\u001b[45m";

        default:
            return "\u001b[30m\u001b[47m";

    }

}

string padNumber(int digits,int n,int empty)
{

    if(empty==0) {

        string ret="";
        digits+=2;

        while (digits--) {
            ret+=" ";
        }

        return ret;
    }

    string s="";

    int padValue = digits - GetNumberOfDigits(n);


    if(n!=1)
    {
        while(n)
        {
            s = char((n%10)+48)+s;
            n/=10;
        }
    }
    else
    {
        s=".";
    }

    if(padValue!=1)
    {
        while(padValue-->0)
        {
            s=" "+s;
            if(padValue!=0)
            {
                s+=" ";
            }
            padValue--;
        }
    }
    else
    {
        s = " "+s;
    }

    return " "+s+" ";

}


unsigned GetNumberOfDigits (int i)
{
    return i > 0 ? (int) log10 ((double) i) + 1 : 1;
}


void drawMatrix(int* matrix)
{

    int pad = 1;

    for(int i=0;i<4;i++){
        int index_i = i*4;
        for(int j=0;j<4;j++){
            pad = max(pad,matrix[index_i+j]);
        }
    }
    int  val = pow(2,pad);

    int digits = GetNumberOfDigits(val);
    digits = digits%2==0?digits+1:digits;

    for(int i=0;i<4;i++)
    {
        int index_i = i*4;
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),0).c_str(),reset.c_str());
        }
        printf("\n");
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),1).c_str(),reset.c_str());
        }
        printf("\n");
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),0).c_str(),reset.c_str());
        }
        printf("\n");
    }

}

void host_moveLeft(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;
        
        int index_j=j*4;
        while (1) {

            mergeCount = 0;

            for (int i = 1; i < 4; i++) {
                if (matrix[index_j+i] != 0 && matrix[index_j+i] == matrix[index_j+i - 1] && temp[i - 1] != 1 && temp[i] != 1) {
                    matrix[index_j+i - 1] = 1 + matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    temp[i - 1] = 1;
                    mergeCount++;
                } else if (matrix[index_j+i] != 0 && matrix[index_j+i - 1] == 0) {
                    matrix[index_j+i - 1] = matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    mergeCount++;
                }
            }

            if (mergeCount == 0) break;

        }
    }

}


void host_moveRight(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;
        int index_j = j*4;

        while (1) {

            mergeCount = 0;

            for (int i = 2; i >= 0; i--) {
                if (matrix[index_j+i] != 0 && matrix[index_j+i] == matrix[index_j+i + 1] && temp[i + 1] != 1 && temp[i] != 1) {
                    matrix[index_j+i + 1] = 1 + matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    temp[i + 1] = 1;
                    mergeCount++;
                } else if (matrix[index_j+i] != 0 && matrix[index_j+i + 1] == 0) {
                    matrix[index_j+i + 1] = matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    mergeCount++;
                }
            }

            if (mergeCount == 0) break;

        }
    }

}


void host_moveUp(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;

        while (1) {

            mergeCount = 0;

            for (int i = 1; i < 4; i++) {
                if (matrix[i*4+j] != 0 && matrix[i*4+j] == matrix[(i - 1)*4+j] && temp[i - 1] != 1 && temp[i] != 1) {
                    matrix[(i - 1)*4+j] = matrix[i*4+j] + 1;
                    matrix[i*4+j] = 0;
                    temp[i - 1] = 1;
                    mergeCount++;
                } else if (matrix[i*4+j] != 0 && matrix[(i - 1)*4+j] == 0) {
                    matrix[(i - 1)*4+j] = matrix[i*4+j];
                    matrix[i*4+j] = 0;
                    mergeCount++;
                }
//               mergeCount--;
            }

            if (mergeCount == 0) break;

        }
    }
}


void host_moveDown(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;

        while (1) {

            mergeCount = 0;

            for (int i = 2; i >= 0 ; i--) {
                if (matrix[i*4+j] != 0 && matrix[i*4+j] == matrix[(i + 1)*4+j] && temp[i + 1] != 1 && temp[i] != 1) {
                    matrix[(i + 1)*4+j] = matrix[i*4+j] + 1;
                    matrix[i*4+j] = 0;
                    temp[i + 1] = 1;
                    mergeCount++;
                } else if (matrix[i*4+j] != 0 && matrix[(i + 1)*4+j] == 0) {
                    matrix[(i + 1)*4+j] = matrix[i*4+j];
                    matrix[i*4+j] = 0;
                    mergeCount++;
                }
//               mergeCount--;
            }

            if (mergeCount == 0) break;

        }
    }
}

void __global__ moveLeft(int* matrix)
{

    int mergeCount;
    int temp[4];
    
    //printf("Hello from GPU..\n");
    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	int start=(4*blockIdx.x)*16+4*threadIdx.x+1;
	int end=(4*blockIdx.x)*16+4*threadIdx.x+4;
	
	//printf("Left Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {
		//printf("Running Left");
        mergeCount = 0;
		//printf("Left Merge Count before: %d \n",mergeCount);
		
        for (int i = start; i < end; i++) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 1] && temp[(i - 1)%4] != 1 && temp[i%4] != 1) {
                matrix[i - 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[(i - 1)%4] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 1] == 0) {
                matrix[i - 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
        if (mergeCount == 0) break;

    }

}


void __global__ moveRight(int* matrix)
{
    int mergeCount;
    int temp[4];

    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	int start=(4*blockIdx.x+1)*16+4*threadIdx.x+2;
	int end=(4*blockIdx.x+1)*16+4*threadIdx.x;
	
	//printf("Right Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {
        mergeCount = 0;
        for (int i = start; i >=end; i--) {
            if (matrix[i] != 0 && matrix[i] == matrix[i + 1] && temp[(i + 1)%4] != 1 && temp[i%4] != 1) {
                matrix[i + 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[(i + 1)%4] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 1] == 0) {
                matrix[i + 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
		//printf("Merge Count outside for: %d \n",mergeCount);
        if (mergeCount == 0) break;

    }
}


void __global__ moveUp(int* matrix)
{

    int mergeCount;
    int temp[16];
    
    for(int i=0;i<16;i++){
        temp[i]=0;
    }
	int start=(threadIdx.x+(4*blockIdx.x+2)*16)+4;
	int end = ((4*blockIdx.x+2)+1)*16;
	//printf("Up Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {

        mergeCount = 0;

        for (int i = start; i < end; i+=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 4] && temp[(i - 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i - 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i - 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 4] == 0) {
                matrix[i - 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }

        if (mergeCount == 0) break;

    }
}


void __global__ moveDown(int* matrix)
{

    int mergeCount;
    int temp[16];

    int start=(((4*blockIdx.x+3)+1)*16)-8+threadIdx.x;
	
	for(int i=0;i<16;i++){
        temp[i]=0;
    }
		
	int end = (4*blockIdx.x+3)*16;
	
	//printf("Down Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {

        mergeCount = 0;

        for (int i = start; i >= end ; i-=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i + 4] && temp[(i + 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i + 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i + 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 4] == 0) {
                matrix[i + 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }

        if (mergeCount == 0) break;

    }
}


void printMatrix(int*** matrix)
{
    for(int i=0;i<4;i++) {
        for (int j = 0; j < 4; j++) {

            if(matrix[i][j]!=0)
                cout<<pow(2,matrix[0][i][j])<<" ";
            else
                cout<<0<<" ";

        }
        cout<<endl;
    }
}

void __global__ generate_copy_for_child_type1(int* input_matrix,int* output_matrix, int* output_rv, int n){
	  int thread_index = threadIdx.x;
	  int block_index = blockIdx.x;
	  int start=((block_index)*16);
      for(int child=0;child<4;child++){
          int child_start = ((block_index*4)+child)*16;
          output_matrix[child_start+thread_index]=input_matrix[start+thread_index];
      }
      output_rv[block_index]=0;
}

void __global__ consolidate_return_values(int* input_rv,int* output_rv,int n){
		int block_index = blockIdx.x;
      int i = 4*block_index;
      output_rv[block_index] = max(max(input_rv[i],input_rv[i+1]),max(input_rv[i+2],input_rv[i+3]));
}

__global__ void runExpectiMaxSearch(int* matrix,int n,int type,int level,int* rv)
{

    int max_level = 7;
	
    if(level == max_level)
    {
        int i=blockIdx.x;
		    //printf("Terminal......... level %d n %d rv %d threadId %d\n",level,n,rv[i],i);
    		if(i==0){
    			printf("Terminal State..");
    		}
        if(i<n){
            rv[i]=0;
            int index_i = i*16;
            int max_tile=0;
            int penalty = 0;
            int free_tiles=0;
            for(int j=0;j<4;j++){
                int index_j = j*4;
                for(int k=0;k<4;k++){
                    rv[i]+=(powf(2,matrix[index_i+index_j+k])*(6-(j+k)));
                    
                    /*if(max_tile < matrix[index_i+index_j+k]){
                        max_tile=matrix[index_i+index_j+k];
                    }*/
                    
                    if (j - 1 >= 0) {
                        penalty += powf(2, abs(matrix[index_i+index_j+k] - matrix[index_i+index_j+k-4]));
                    }
                    if (j + 1 <= 3) {
                        penalty += powf(2, abs(matrix[index_i+index_j+k] - matrix[index_i+index_j+k+4]));
                    }
                    if (k - 1 >= 0) {
                        penalty += powf(2, abs(matrix[index_i+index_j+k] - matrix[index_i+index_j+k-1]));
                    }
                    if (k + 1 <= 3) {
                        penalty += powf(2, abs(matrix[index_i+index_j+k] - matrix[index_i+index_j+k+1]));
                    }
                    
                    if(matrix[index_i+index_j+k]==0){
                        free_tiles++;
                    }
                }
            }
            /*if(max_tile==matrix[index_i]){
                rv[i]+=1000;
            }*/
            rv[i]-=penalty;
            rv[i]+=(free_tiles*100);
        }
    }

    else if(type == 1)//Error in move left or right.. needs to be discovered..
    {
        //LRUD configuration
        printf("Directions......... level %d n %d \n",level,n);
        int* child_matrix = (int *)malloc(n*4*16*sizeof(int));
        int* child_rv = (int *)malloc(n*4*sizeof(int));
    		if(child_matrix==NULL || child_rv==NULL){
    			printf("No enough memory to allocate for matrix.. %x %x\n",child_matrix,child_rv);
          free(child_matrix);
          free(child_rv);
    			runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv);
    			hipDeviceSynchronize();
    			return;
    		}
        generate_copy_for_child_type1<<<n,16>>>((int *)matrix,child_matrix,child_rv,n);
        hipDeviceSynchronize();
		
		    printf("n= %d \n",n);

        hipStream_t left;
        hipStreamCreateWithFlags(&left, hipStreamNonBlocking);
        moveLeft<<<n,4,0,left>>>(child_matrix);//should always be called with 4 threads, each for a row and a block for a matrix
		
        hipStream_t right;
        hipStreamCreateWithFlags(&right, hipStreamNonBlocking);
        moveRight<<<n,4,0,right>>>(child_matrix);
        
        hipStream_t up;
        hipStreamCreateWithFlags(&up, hipStreamNonBlocking);
        moveUp<<<n,4,0,up>>>(child_matrix);
        
        hipStream_t down;
        hipStreamCreateWithFlags(&down, hipStreamNonBlocking);
        moveDown<<<n,4,0,down>>>(child_matrix);
        hipDeviceSynchronize();

    		if(level!=max_level-1){
    			runExpectiMaxSearch<<<1,1>>>(child_matrix,4*n,type+1,level+1,child_rv);
    		}
    		else{
    			runExpectiMaxSearch<<<4*n,1>>>(child_matrix,4*n,type+1,level+1,child_rv);
    		}
    		hipDeviceSynchronize();
        consolidate_return_values<<<n,1>>>(child_rv,rv,n);
        hipDeviceSynchronize();
        if(level==0){
            if(child_rv[0]>child_rv[1]){
                if(child_rv[0]>child_rv[2]){
                    if(child_rv[0]>child_rv[3]){
                        rv[0]=0;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
            else{
                if(child_rv[1]>child_rv[2]){
                    if(child_rv[1]>child_rv[3]){
                        rv[0]=1;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
        }
        free(child_rv);
		free(child_matrix);
		printf("5.........\n");
    }
    else {
        printf("Chances.........level %d n %d \n",level,n);
        int count = 0;
    		int *n_count;
    		n_count = (int *)malloc(sizeof(int)*n);
        int *temp;
        temp = (int *)malloc(sizeof(int)*n);
    		if(n_count==NULL || temp==NULL){
          free(n_count);
          free(temp);
          printf("No enough memory to allocate for chance nodes.. \n");
          runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv);
          hipDeviceSynchronize();
          return;
    		}

        for (int i = 0; i < n; i++) {
      			int index_i = i*16;
      			int num=0;
      			temp[i]=0;
      			n_count[i]=0;
			
            for (int j = 0; j < 4; j++) {
				        int index_j = j*4;
                for (int k = 0; k < 4; k++) {
                    if (matrix[index_i+index_j+k] == 0){
                        count++;
          						  num++;
					          }
                }
            }

            if (i == 0){
                temp[0] = count;
        				n_count[0] = count;
            }
            else{
                temp[i] = num;
                n_count[i] = count;
            }
        }
        int *matrix1 = (int *)malloc(count*16*(sizeof(int)));

        int *rv1 = (int *)malloc(count*sizeof(int));
    		if(matrix1==NULL || rv1==NULL){
         free(matrix1);
         free(rv1);
    			printf("No enough memory to allocate for chance nodes @.. \n");
    			runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv);
    			hipDeviceSynchronize();
    			return;
    		}

        for (int i = 0; i < count; i++) rv1[i] = 0;

		int k=0;
		
        for (int l = 0; l < n; l++) {
            int flag=0;
			int index_l = l*16;
            for (int i = 0; i < temp[l]; i++) {
                int index = i;
                if (l != 0) {
                    index += n_count[l - 1];
                }
                flag=0;
				int index_i = index*16;
                for (int j = 0; j < 4; j++) {
					int index_j = j*4;
                    for (int k = 0; k < 4; k++) {
                        if (matrix[index_l+index_j+k] == 0 && flag==0) {
                            matrix1[index_i+index_j+k] = 1;
                            matrix[index_l+index_j+k] = -1;
                            flag=1;
                        } else {
                            matrix1[index_i+index_j+k] = matrix[index_l+index_j+k];
                            if(matrix[index_l+index_j+k] == -1){
                                matrix1[index_i+index_j+k] = 0;
                            }
                        }

                    }
                }

            }

        }

    free(n_count);
		if(level!=max_level-1){
			runExpectiMaxSearch<<<1,1>>>(matrix1, count, type-1, level + 1, rv1);
		}
		else{
			runExpectiMaxSearch<<<count,1>>>(matrix1, count, type-1, level + 1, rv1);
		}

		hipDeviceSynchronize();
		int p;
		k=p=0;
        for (int l = 0; l < n; l++) {
            int q= temp[l];
			if(q==0){
				rv[k]=0;
				k++;
				continue;
			}
			while(q--){
				rv[k]+=rv1[p++];
			}
			rv[k]/=temp[l];
			k++;
        }
		free(matrix1);
		free(temp);
		free(rv1);
    }

}

int placerandomtile(int* matrix){
    int place=0;
    int random_pos[16];
    for(int i=0;i<4;i++){
        int index_i=i*4;
        for(int j=0;j<4;j++){
            if(matrix[index_i+j]==0){
                random_pos[place]=i*4+j;
                place++;
            }
        }
    }
    if(place==0){
        return 1;
    }
    int random = rand()%place;
    place = random_pos[random];
    int x = place/4;
    int y = place%4;
    //int tiles[10] = {1,1,1,1,1,1,1,1,2,2};
    random = rand()%10;
    matrix[4*x+y]=1;//tiles[random];
    return 0;
}

int main() {

    int* matrix = (int *)malloc(sizeof(int)*16);
  	int* d_matrix;
      int* rv = (int *)malloc(sizeof(int));
  	int *d_rv;
  
  	int matrix_size = sizeof(int)*16;
  	int rv_size = sizeof(int)*1;
	
    rv[0]=0;

    matrix[0] = 0;
    matrix[1] = 0;
    matrix[2] = 2;
    matrix[3] = 1;

    matrix[4] = 0;
    matrix[5] = 0;
    matrix[6] = 0;
    matrix[7] = 0;

    matrix[8] = 0;
    matrix[9] = 0;
    matrix[10] = 0;
    matrix[11] = 0;

    matrix[12] = 0;
    matrix[13] = 0;
    matrix[14] = 0;
    matrix[15] = 0;
	
    cout<<"Boooooom"<<endl;
    
    hipMalloc(&d_matrix,matrix_size);
 	  hipMalloc(&d_rv,rv_size);
    
    int times=1000;
    while(times--) {

        cout<<times<<endl;
        drawMatrix(matrix);

        hipMemcpy(d_matrix,matrix,matrix_size,hipMemcpyHostToDevice);
      	hipMemcpy(d_rv,rv,rv_size,hipMemcpyHostToDevice);
      	
      	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 8);
      	runExpectiMaxSearch<<<1,1>>>(d_matrix,1,1,0,d_rv);
      	
      	hipDeviceSynchronize();
      	hipError_t err;
      	err = hipGetLastError();
      	if(err!=hipSuccess){
      		printf("Error %s \n",hipGetErrorString(err));
         exit(1);
      	}
      	//hipMemcpy(matrix,d_matrix,matrix_size,hipMemcpyDeviceToHost);
      	hipMemcpy(rv,d_rv,rv_size,hipMemcpyDeviceToHost);
      	hipDeviceSynchronize();

        printf("%d ",rv[0]);
        if (rv[0] == 0) {
            cout << "Left" << endl;
            host_moveLeft(matrix);
            drawMatrix(matrix);
            cout << endl;

        }
        if (rv[0] == 1) {
            cout << "Right" << endl;
            host_moveRight(matrix);
            drawMatrix(matrix);
            cout << endl;
        }
        if (rv[0] == 2) {
            cout << "Up" << endl;
            host_moveUp(matrix);
            drawMatrix(matrix);
            cout << endl;
        }
        if (rv[0] == 3) {
            cout << "Down" << endl;
            host_moveDown(matrix);
            drawMatrix(matrix);
            cout << endl;
        }

        if(placerandomtile(matrix)==1)
            break;
    }

    hipFree(d_matrix);
    hipFree(d_rv);
    free(matrix);
    free(rv);
    return 0;
}

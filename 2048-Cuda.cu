#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <random>
#include <queue>
#include <sys/types.h>
#include<chrono>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

string reset = "\u001b[0m";

unsigned GetNumberOfDigits (int i);

double board_score=0;

string getColor(int value)
{
    switch(value%10)
    {

        case 1:
            return "\u001b[37m\u001b[43;1m";

        case 2:
            return "\u001b[37m\u001b[44;1m";

        case 3:
            return "\u001b[37m\u001b[45;1m";

        case 4:
            return "\u001b[37m\u001b[46;1m";

        case 5:
            return "\u001b[37m\u001b[40;1m";

        case 6:
            return "\u001b[37m\u001b[41m";

        case 7:
            return "\u001b[37m\u001b[41;1m";

        case 8:
            return "\u001b[37m\u001b[42;1m";

        case 9:
            return "\u001b[37m\u001b[42m";

//        case 0:
//            return "\u001b[37m\u001b[45m";

        default:
            return "\u001b[30m\u001b[47m";

    }

}

string padNumber(int digits,int n,int empty)
{

    if(empty==0) {

        string ret="";
        digits+=2;

        while (digits--) {
            ret+=" ";
        }

        return ret;
    }

    string s="";

    int padValue = digits - GetNumberOfDigits(n);


    if(n!=1)
    {
        while(n)
        {
            s = char((n%10)+48)+s;
            n/=10;
        }
    }
    else
    {
        s=".";
    }

    if(padValue!=1)
    {
        while(padValue-->0)
        {
            s=" "+s;
            if(padValue!=0)
            {
                s+=" ";
            }
            padValue--;
        }
    }
    else
    {
        s = " "+s;
    }

    return " "+s+" ";

}


unsigned GetNumberOfDigits (int i)
{
    return i > 0 ? (int) log10 ((double) i) + 1 : 1;
}
//all the above funcitons are used as helper functions for the drawmatrix funciton.
//this function is used to display the matrix on the console, this can be used only on linux based systems.
void drawMatrix(int* matrix)
{

    int pad = 1;

    for(int i=0;i<4;i++){
        int index_i = i*4;
        for(int j=0;j<4;j++){
            pad = max(pad,matrix[index_i+j]);
        }
    }
    int  val = pow(2,pad);

    int digits = GetNumberOfDigits(val);
    digits = digits%2==0?digits+1:digits;

    for(int i=0;i<4;i++)
    {
        int index_i = i*4;
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),0).c_str(),reset.c_str());
        }
        printf("\n");
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),1).c_str(),reset.c_str());
        }
        printf("\n");
        for(int j=0;j<4;j++)
        {
            printf("%s%s%s",getColor(matrix[index_i+j]).c_str(),padNumber(digits,pow(2,matrix[index_i+j]),0).c_str(),reset.c_str());
        }
        printf("\n");
    }

}

//this is function used in main to perform the left operation on the node, this function also takes of increasin the score when there are merges
void host_moveLeft(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;
        
        int index_j=j*4;
        while (1) {

            mergeCount = 0;

            for (int i = 1; i < 4; i++) {
                if (matrix[index_j+i] != 0 && matrix[index_j+i] == matrix[index_j+i - 1] && temp[i - 1] != 1 && temp[i] != 1) {
                    matrix[index_j+i - 1] = 1 + matrix[index_j+i];
					board_score+=powf(2,matrix[index_j+i - 1]);
                    matrix[index_j+i] = 0;
                    temp[i - 1] = 1;
                    mergeCount++;
                } else if (matrix[index_j+i] != 0 && matrix[index_j+i - 1] == 0) {
                    matrix[index_j+i - 1] = matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    mergeCount++;
                }
            }

            if (mergeCount == 0) break;

        }
    }

}

//this is function used in main to perform the right operation on the node, this function also takes of increasin the score when there are merges
void host_moveRight(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;
        int index_j = j*4;

        while (1) {

            mergeCount = 0;

            for (int i = 2; i >= 0; i--) {
                if (matrix[index_j+i] != 0 && matrix[index_j+i] == matrix[index_j+i + 1] && temp[i + 1] != 1 && temp[i] != 1) {
                    matrix[index_j+i + 1] = 1 + matrix[index_j+i];
					board_score+=powf(2,matrix[index_j+i +1]);
                    matrix[index_j+i] = 0;
                    temp[i + 1] = 1;
                    mergeCount++;
                } else if (matrix[index_j+i] != 0 && matrix[index_j+i + 1] == 0) {
                    matrix[index_j+i + 1] = matrix[index_j+i];
                    matrix[index_j+i] = 0;
                    mergeCount++;
                }
            }

            if (mergeCount == 0) break;

        }
    }

}

//this is function used in main to perform the up operation on the node, this function also takes of increasin the score when there are merges
void host_moveUp(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;

        while (1) {

            mergeCount = 0;

            for (int i = 1; i < 4; i++) {
                if (matrix[i*4+j] != 0 && matrix[i*4+j] == matrix[(i - 1)*4+j] && temp[i - 1] != 1 && temp[i] != 1) {
                    matrix[(i - 1)*4+j] = matrix[i*4+j] + 1;
					board_score+=powf(2,matrix[(i - 1)*4+j]);
                    matrix[i*4+j] = 0;
                    temp[i - 1] = 1;
                    mergeCount++;
                } else if (matrix[i*4+j] != 0 && matrix[(i - 1)*4+j] == 0) {
                    matrix[(i - 1)*4+j] = matrix[i*4+j];
                    matrix[i*4+j] = 0;
                    mergeCount++;
                }
//               mergeCount--;
            }

            if (mergeCount == 0) break;

        }
    }
}

//this is function used in main to perform the down operation on the node, this function also takes of increasin the score when there are merges
void host_moveDown(int* matrix)
{

    int mergeCount;
    int temp[4];

    for(int j=0;j<4;j++) {

        temp[0] = temp[1] = temp[2] = temp[3] = 0;

        while (1) {

            mergeCount = 0;

            for (int i = 2; i >= 0 ; i--) {
                if (matrix[i*4+j] != 0 && matrix[i*4+j] == matrix[(i + 1)*4+j] && temp[i + 1] != 1 && temp[i] != 1) {
                    matrix[(i + 1)*4+j] = matrix[i*4+j] + 1;
					board_score+=powf(2,matrix[(i + 1)*4+j]);
                    matrix[i*4+j] = 0;
                    temp[i + 1] = 1;
                    mergeCount++;
                } else if (matrix[i*4+j] != 0 && matrix[(i + 1)*4+j] == 0) {
                    matrix[(i + 1)*4+j] = matrix[i*4+j];
                    matrix[i*4+j] = 0;
                    mergeCount++;
                }
//               mergeCount--;
            }

            if (mergeCount == 0) break;

        }
    }
}

//this function is used to perform move left operation on all the rows of all the nodes parallely. No. of blocks = number of matrices, threads = 4, each thread for each row
void __global__ moveLeft(int* matrix)
{

    int mergeCount;
    int temp[4];
    
    //printf("Hello from GPU..\n");
    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	int start=(4*blockIdx.x)*16+4*threadIdx.x+1;
	int end=(4*blockIdx.x)*16+4*threadIdx.x+4;
	
	//printf("Left Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {
		//printf("Running Left");
        mergeCount = 0;
		//printf("Left Merge Count before: %d \n",mergeCount);
		
        for (int i = start; i < end; i++) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 1] && temp[(i - 1)%4] != 1 && temp[i%4] != 1) {
                matrix[i - 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[(i - 1)%4] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 1] == 0) {
                matrix[i - 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
        if (mergeCount == 0) break;

    }

}

//this function is used to perform move left operation on all the rows of all the nodes parallely. No. of blocks = number of matrices, threads = 4, each thread for each row
void __global__ moveRight(int* matrix)
{
    int mergeCount;
    int temp[4];

    temp[0] = temp[1] = temp[2] = temp[3] = 0;
	int start=(4*blockIdx.x+1)*16+4*threadIdx.x+2;
	int end=(4*blockIdx.x+1)*16+4*threadIdx.x;
	
	//printf("Right Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {
        mergeCount = 0;
        for (int i = start; i >=end; i--) {
            if (matrix[i] != 0 && matrix[i] == matrix[i + 1] && temp[(i + 1)%4] != 1 && temp[i%4] != 1) {
                matrix[i + 1] = 1 + matrix[i];
                matrix[i] = 0;
                temp[(i + 1)%4] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 1] == 0) {
                matrix[i + 1] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }
		//printf("Merge Count outside for: %d \n",mergeCount);
        if (mergeCount == 0) break;

    }
}

//this function is used to perform move up operation on all the columns of all the nodes parallely. No. of blocks = number of matrices, threads = 4, each thread for each column
void __global__ moveUp(int* matrix)
{

    int mergeCount;
    int temp[16];
    
    for(int i=0;i<16;i++){
        temp[i]=0;
    }
	int start=(threadIdx.x+(4*blockIdx.x+2)*16)+4;
	int end = ((4*blockIdx.x+2)+1)*16;
	//printf("Up Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {

        mergeCount = 0;

        for (int i = start; i < end; i+=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i - 4] && temp[(i - 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i - 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i - 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i - 4] == 0) {
                matrix[i - 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }

        if (mergeCount == 0) break;

    }
}

//this function is used to perform move down operation on all the columns of all the nodes parallely. No. of blocks = number of matrices, threads = 4, each thread for each column
void __global__ moveDown(int* matrix)
{

    int mergeCount;
    int temp[16];

    int start=(((4*blockIdx.x+3)+1)*16)-8+threadIdx.x;
	
	for(int i=0;i<16;i++){
        temp[i]=0;
    }
		
	int end = (4*blockIdx.x+3)*16;
	
	//printf("Down Thread Idx: %d ,Block Idx: %d, start: %d , end: %d , gridDIM: %d \n", threadIdx.x,blockIdx.x,start,end,gridDim.x);
    while (1) {

        mergeCount = 0;

        for (int i = start; i >= end ; i-=4) {
            if (matrix[i] != 0 && matrix[i] == matrix[i + 4] && temp[(i + 4)%16] != 1 && temp[i%16] != 1) {
                matrix[i + 4] = matrix[i] + 1;
                matrix[i] = 0;
                temp[(i + 4)%16] = 1;
                mergeCount++;
            } else if (matrix[i] != 0 && matrix[i + 4] == 0) {
                matrix[i + 4] = matrix[i];
                matrix[i] = 0;
                mergeCount++;
            }
        }

        if (mergeCount == 0) break;

    }
}


void printMatrix(int*** matrix)
{
    for(int i=0;i<4;i++) {
        for (int j = 0; j < 4; j++) {

            if(matrix[i][j]!=0)
                cout<<pow(2,matrix[0][i][j])<<" ";
            else
                cout<<0<<" ";

        }
        cout<<endl;
    }
}

//this function is used to generate child copies in parallel for all the input matrices.
void __global__ generate_copy_for_child_type1(int* input_matrix,int* output_matrix, int* output_rv, int n){
	  int thread_index = threadIdx.x;
	  int block_index = blockIdx.x;
	  int start=((block_index)*16);
      for(int child=0;child<4;child++){
          int child_start = ((block_index*4)+child)*16;
          output_matrix[child_start+thread_index]=input_matrix[start+thread_index];
      }
      output_rv[block_index]=0;
}

//this function is used to consolidate the return values from the child parallely and decide which action to taken based on the value store in rv[].
void __global__ consolidate_return_values(int* input_rv,int* output_rv,int n){
		int block_index = blockIdx.x;
      int i = 4*block_index;
      output_rv[block_index] = max(max(input_rv[i],input_rv[i+1]),max(input_rv[i+2],input_rv[i+3]));
}

//this is the core function which does the expecti max search
__global__ void runExpectiMaxSearch(int* matrix,int n,int type,int level,int* rv)
{

    int max_level = 7;
	
    if(level == max_level)  //If this is the terminal value, scores of the terminal states will be calculated.
    {
        int i=blockIdx.x;
		    //printf("Terminal......... level %d n %d rv %d threadId %d\n",level,n,rv[i],i);
    		if(i==0){
    			//printf("Terminal State..");
    		}
		//double weights[16] = {10,8,7,6.5,.5,.7,1,3,-.5,-1.5,-1.8,-2,-3.8,-3.7,-3.5,-3};
		//double weights[16] = {32,8,4,2,8,4,2,0,4,2,0,0,2,0,0,0};
		double score = 0;
        if(i<n){
            rv[i]=0;
            int index_i = i*16;
            int max_tile=0;
            int penalty = 0;
            int free_tiles=0;
            for(int j=0;j<4;j++){
                int index_j = j*4;
                for(int k=0;k<4;k++){
                    rv[i]+=(powf(2,matrix[index_i+index_j+k])*(6-(j+k)));
                    //score+=(powf(2,matrix[index_i+index_j+k])*(weights[index_j+k]));
                    if(max_tile < matrix[index_i+index_j+k]){
                        max_tile=matrix[index_i+index_j+k];
                    }
                    
                    if (j - 1 >= 0) {
                        penalty += (powf(2,matrix[index_i+index_j+k]) - powf(2,matrix[index_i+index_j+k-4]));
                    }
                    if (j + 1 <= 3) {
                        penalty += (powf(2,matrix[index_i+index_j+k]) - powf(2,matrix[index_i+index_j+k+4]));
                    }
                    if (k - 1 >= 0) {
                        penalty += (powf(2,matrix[index_i+index_j+k]) - powf(2,matrix[index_i+index_j+k-1]));
                    }
                    if (k + 1 <= 3) {
                        penalty += (powf(2,matrix[index_i+index_j+k]) - powf(2,matrix[index_i+index_j+k+1]));
                    }
                    
                    if(matrix[index_i+index_j+k]==0){
                        free_tiles++;
                    }
                }
            }
			
            /*if(max_tile==matrix[index_i]){
                rv[i]+=1000;
            }
			if(free_tiles<=2){
				rv[i]-=500;
			}*/
			/*if(free_tiles==0){
				rv[i]=-2000000;
			}*/
            rv[i]-=penalty;
		
        }
    }

    else if(type == 1)//If this is the max node, generate its children and call the chance node
    {
        //LRUD configuration
        //printf("Directions......... level %d n %d \n",level,n);
        int* child_matrix = (int *)malloc(n*4*16*sizeof(int));
        int* child_rv = (int *)malloc(n*4*sizeof(int));
    		if(child_matrix==NULL || child_rv==NULL){
    			//printf("No enough memory to allocate for matrix.. %x %x\n",child_matrix,child_rv);
          free(child_matrix);
          free(child_rv);
    			runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv); //If extra memory cannot be allocated, call the terminal state. This is bcz of system constraints on my laptop.
    			hipDeviceSynchronize(); //synchronization
    			return;
    		}
        generate_copy_for_child_type1<<<n,16>>>((int *)matrix,child_matrix,child_rv,n); //generating 4 copies for each matrix that is input.
        hipDeviceSynchronize();
		
		    //printf("n= %d \n",n);

        hipStream_t left;
        hipStreamCreateWithFlags(&left, hipStreamNonBlocking);
        moveLeft<<<n,4,0,left>>>(child_matrix);//should always be called with 4 threads, each for a row and a block for a matrix
		
        hipStream_t right;
        hipStreamCreateWithFlags(&right, hipStreamNonBlocking);
        moveRight<<<n,4,0,right>>>(child_matrix);//generated right children for all the input matrices parallely
        
        hipStream_t up;
        hipStreamCreateWithFlags(&up, hipStreamNonBlocking);
        moveUp<<<n,4,0,up>>>(child_matrix);//generated up children for all the input matrices parallely
        
        hipStream_t down;
        hipStreamCreateWithFlags(&down, hipStreamNonBlocking);
        moveDown<<<n,4,0,down>>>(child_matrix);//generated down children for all the input matrices parallely
        hipDeviceSynchronize(); //Launched all the kernels in parallel and all the left, right, up, down happen in parallel and this here we stop to synchronize

    		if(level!=max_level-1){
    			runExpectiMaxSearch<<<1,1>>>(child_matrix,4*n,type+1,level+1,child_rv);
    		}
    		else{
    			runExpectiMaxSearch<<<4*n,1>>>(child_matrix,4*n,type+1,level+1,child_rv);//calling the next chance node.
    		}
    		hipDeviceSynchronize();
        consolidate_return_values<<<n,1>>>(child_rv,rv,n);//consolidating the return values based on the valued returned from chance node.
        hipDeviceSynchronize();
        if(level==0){
            if(child_rv[0]>child_rv[1]){
                if(child_rv[0]>child_rv[2]){
                    if(child_rv[0]>child_rv[3]){
                        rv[0]=0;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
            else{
                if(child_rv[1]>child_rv[2]){
                    if(child_rv[1]>child_rv[3]){
                        rv[0]=1;
                    }
                    else{
                        rv[0]=3;
                    }
                }
                else{
                    if(child_rv[2]>child_rv[3]){
                        rv[0]=2;
                    }
                    else{
                        rv[0]=3;
                    }
                }
            }
        }
        free(child_rv);
		free(child_matrix);
		//printf("5.........\n");
    }
    else { //This is the chance node, where the children are generated by placing a random tile at each empty tile and calls the max node.
        //printf("Chances.........level %d n %d \n",level,n);
        int count = 0;
    		int *n_count;
    		n_count = (int *)malloc(sizeof(int)*n);
        int *temp;
        temp = (int *)malloc(sizeof(int)*n);
    		if(n_count==NULL || temp==NULL){
          free(n_count);
          free(temp);
          //printf("No enough memory to allocate for chance nodes.. \n");
          runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv); //if cannot allocate extra memory, calling the terminal state rightaway, bcz of my system constraints
          hipDeviceSynchronize();
          return;
    		}

        for (int i = 0; i < n; i++) {
      			int index_i = i*16;
      			int num=0;
      			temp[i]=0;
      			n_count[i]=0;
			
            for (int j = 0; j < 4; j++) {
				        int index_j = j*4;
                for (int k = 0; k < 4; k++) {
                    if (matrix[index_i+index_j+k] == 0){
                        count++;
          						  num++;
					          }
                }
            }

            if (i == 0){
                temp[0] = count;
        				n_count[0] = count;
            }
            else{
                temp[i] = num;
                n_count[i] = count;
            }
        }
        int *matrix1 = (int *)malloc(count*16*(sizeof(int)));

        int *rv1 = (int *)malloc(count*sizeof(int));
    		if(matrix1==NULL || rv1==NULL){
         free(matrix1);
         free(rv1);
    			//printf("No enough memory to allocate for chance nodes @.. \n");
    			runExpectiMaxSearch<<<n,1>>>(matrix,n,type+1,max_level,rv);//if cannot allocate extra memory, calling the terminal state rightaway, bcz of my system constraints
    			hipDeviceSynchronize();
    			return;
    		}

        for (int i = 0; i < count; i++) rv1[i] = 0;

		int k=0;
		
        for (int l = 0; l < n; l++) { //creating copies(children) with placing a random tile at free tile
            int flag=0;
			int index_l = l*16;
            for (int i = 0; i < temp[l]; i++) {
                int index = i;
                if (l != 0) {
                    index += n_count[l - 1];
                }
                flag=0;
				int index_i = index*16;
                for (int j = 0; j < 4; j++) {
					int index_j = j*4;
                    for (int k = 0; k < 4; k++) {
                        if (matrix[index_l+index_j+k] == 0 && flag==0) {
                            matrix1[index_i+index_j+k] = 1;
                            matrix[index_l+index_j+k] = -1;
                            flag=1;
                        } else {
                            matrix1[index_i+index_j+k] = matrix[index_l+index_j+k];
                            if(matrix[index_l+index_j+k] == -1){
                                matrix1[index_i+index_j+k] = 0;
                            }
                        }

                    }
                }

            }

        }

    free(n_count);
		if(level!=max_level-1){
			runExpectiMaxSearch<<<1,1>>>(matrix1, count, type-1, level + 1, rv1);
		}
		else{
			runExpectiMaxSearch<<<count,1>>>(matrix1, count, type-1, level + 1, rv1);//calling the next state which is max node.
		}

		hipDeviceSynchronize(); //synchronization
		int p;
		k=p=0;
        for (int l = 0; l < n; l++) { //consolidate_return_values
            int q= temp[l];
			if(q==0){
				rv[k]=0;
				k++;
				continue;
			}
			while(q--){
				rv[k]+=rv1[p++];
			}
			rv[k]/=temp[l];
			k++;
        }
		free(matrix1);
		free(temp);
		free(rv1);
    }

}

//this function is used to place a random tile using the random number generator
int placerandomtile(int* matrix){

	std::random_device rd;
    std::mt19937 mt(rd()); //using a random device generator to generate random position and place a tile there.
	
    int place=0;
    int random_pos[16];
    for(int i=0;i<4;i++){
        int index_i=i*4;
        for(int j=0;j<4;j++){
            if(matrix[index_i+j]==0){
                random_pos[place]=i*4+j;
                place++;
            }
        }
    }
	
    if(place==0){
        return 1;
    }
	std::uniform_real_distribution<double> dist(0.0, place);
    int random = ((int)dist(mt))%place;
    place = random_pos[random];
    int x = place/4;
    int y = place%4;
    //int tiles[10] = {1,1,1,1,1,1,1,1,2,2};
    random = rand()%10;
    matrix[4*x+y]=1;//tiles[random];
    return 0;
}

int main() {

    int* matrix = (int *)malloc(sizeof(int)*16);
  	int* d_matrix;
      int* rv = (int *)malloc(sizeof(int));
  	int *d_rv;
  
  	int matrix_size = sizeof(int)*16;
  	int rv_size = sizeof(int)*1;
	
    rv[0]=0;

    matrix[0] = 0;
    matrix[1] = 0;
    matrix[2] = 0;
    matrix[3] = 0;

    matrix[4] = 0;
    matrix[5] = 0;
    matrix[6] = 0;
    matrix[7] = 0;

    matrix[8] = 0;
    matrix[9] = 0;
    matrix[10] = 0;
    matrix[11] = 0;

    matrix[12] = 0;
    matrix[13] = 0;
    matrix[14] = 1;
    matrix[15] = 1;
	
    cout<<"Boooooom"<<endl;
    
    hipMalloc(&d_matrix,matrix_size);
 	  hipMalloc(&d_rv,rv_size);
    
	using namespace std::chrono;
    high_resolution_clock::time_point t1;
    high_resolution_clock::time_point t2;
	
    int times=5000;
	double total_time = 0;
    while(times--) {

        cout<<times<<endl;
        drawMatrix(matrix);

        hipMemcpy(d_matrix,matrix,matrix_size,hipMemcpyHostToDevice);
      	hipMemcpy(d_rv,rv,rv_size,hipMemcpyHostToDevice);
      	
      	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 8); //For now, setting the max recursive synchronization depth to 8, as because of system constraints cannot go beyond level 7.
		t1 = high_resolution_clock::now();
      	runExpectiMaxSearch<<<1,1>>>(d_matrix,1,1,0,d_rv); //calling the search, will get the value in d_rv which will be used to make the next move.
      	
      	hipDeviceSynchronize();
      	hipError_t err;
      	err = hipGetLastError();
      	if(err!=hipSuccess){
      		printf("Error %s \n",hipGetErrorString(err));
         exit(1);
      	}
      	//hipMemcpy(matrix,d_matrix,matrix_size,hipMemcpyDeviceToHost);
      	hipMemcpy(rv,d_rv,rv_size,hipMemcpyDeviceToHost); //getting back the value from device to host.
      	hipDeviceSynchronize();
		t2 = high_resolution_clock::now();
		duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
		//printf("Time Span in GPU: %f",time_span.count());

		total_time+=time_span.count();
        printf("%d ",rv[0]);
        if (rv[0] == 0) {
            cout << "Left" << endl;
            host_moveLeft(matrix);
            drawMatrix(matrix);
            cout << endl;

        }
        if (rv[0] == 1) {
            cout << "Right" << endl;
            host_moveRight(matrix);
            drawMatrix(matrix);
            cout << endl;
        }
        if (rv[0] == 2) {
            cout << "Up" << endl;
            host_moveUp(matrix);
            drawMatrix(matrix);
            cout << endl;
        }
        if (rv[0] == 3) {
            cout << "Down" << endl;
            host_moveDown(matrix);
            drawMatrix(matrix);
            cout << endl;
        }

        if(placerandomtile(matrix)==1){
			printf("Avg time taken to make a move is %f",total_time/(5000-times));
			//printf("Score Scored: %f",board_score);
            break;
		}
    }

    hipFree(d_matrix);
    hipFree(d_rv);
    free(matrix);
    free(rv);
    return 0;
}
